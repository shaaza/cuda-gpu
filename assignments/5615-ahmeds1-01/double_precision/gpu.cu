#include "hip/hip_runtime.h"
#include <stdio.h>
#include "shared.h"
#include "block_size.h"

void cuda_last_error_check (const char *message);

// Add rows kernel & related operations
__global__ void add_rows_gpu_kernel(double* mat, double* out, int n, int m);
void add_rows_gpu(double* rowsum, double* mat1d, int n, int m, struct Timer* timer);

// Add columns kernel & related operations
__global__ void add_cols_gpu_kernel(double* mat, double* out, int n, int m);
void add_columns_gpu(double* rowsum, double* mat1d, int n, int m, struct Timer* timer);

// Reduce vector kernel & related operations
__global__ void reduce_vector_gpu_kernel(double* vec, double* result, int n);
void reduce_vector_gpu(double* vec, double* result, int n, struct Timer* timer);

extern struct Options options; // Global config var

void perform_gpu_operations(double* mat1d, struct Stats* stats) {
  int n = options.rows;
  int m = options.cols;

  double* rowsum = (double*) malloc(n*sizeof(double));
  add_rows_gpu(rowsum, mat1d, n, m, &(stats->add_rows));

  double* colsum = (double*) malloc(n*sizeof(double));
  add_columns_gpu(colsum, mat1d, n, m, &(stats->add_columns));

  double rowsum_reduced;
  reduce_vector_gpu(rowsum, &rowsum_reduced, n, &(stats->reduce_vector_rows));

  double colsum_reduced;
  reduce_vector_gpu(colsum, &colsum_reduced, m, &(stats->reduce_vector_cols));

  print_compute_results((char*) "GPU Results:", rowsum, colsum, rowsum_reduced, colsum_reduced, n, m);

  // Free memory
  free(rowsum);
  free(colsum);
}

void add_rows_gpu(double* rowsum, double* mat1d, int n, int m, struct Timer* timer) {
  // Compute execution GPU config
  dim3 dimBlock(BLOCK_SIZE, 1);
  int blocks_in_grid = (int) ceil((double) n / BLOCK_SIZE);
  dim3 dimGrid(blocks_in_grid, 1);

  // Device: alloc
  double* mat1d_GPU;
  double* rowsum_GPU;
  hipMalloc((void**) &mat1d_GPU, n*m*sizeof(double));
  hipMalloc((void**) &rowsum_GPU, n*sizeof(double));

  // Host->Device copy
  hipMemcpy(mat1d_GPU, mat1d, n*m*sizeof(double), hipMemcpyHostToDevice);

  // Device: execution + timing
  start_timer(timer);
  add_rows_gpu_kernel<<<dimGrid, dimBlock>>>(mat1d_GPU, rowsum_GPU, n, m);
  end_timer(timer);

  char err_msg[100];
  sprintf(err_msg, "add_rows_gpu (threads/block: %d, blocks: %d)", BLOCK_SIZE, blocks_in_grid);
  cuda_last_error_check(err_msg);

  // Device->Host copy
  hipMemcpy(rowsum, rowsum_GPU, n*sizeof(double), hipMemcpyDeviceToHost);

  hipFree(mat1d_GPU);
  hipFree(rowsum_GPU);
}

void add_columns_gpu(double* colsum, double* mat1d, int n, int m, struct Timer* timer) {
  // Compute execution GPU config
  dim3 dimBlock(1, BLOCK_SIZE);
  int blocks_in_grid = (int) ceil((double) n / BLOCK_SIZE);
  dim3 dimGrid(blocks_in_grid, 1);

  // Device: alloc
  double* mat1d_GPU;
  double* colsum_GPU;
  hipMalloc((void**) &mat1d_GPU, n*m*sizeof(double));
  hipMalloc((void**) &colsum_GPU, m*sizeof(double));

  // Host->Device copy
  hipMemcpy(mat1d_GPU, mat1d, n*m*sizeof(double), hipMemcpyHostToDevice);

  // Device: execution + timing
  start_timer(timer);
  add_cols_gpu_kernel<<<dimGrid, dimBlock>>>(mat1d_GPU, colsum_GPU, n, m);
  end_timer(timer);

  char err_msg[100];
  sprintf(err_msg, "add_columns_gpu (threads/block: %d, blocks: %d)", BLOCK_SIZE, blocks_in_grid);
  cuda_last_error_check(err_msg);

  // Device->Host copy
  hipMemcpy(colsum, colsum_GPU, m*sizeof(double), hipMemcpyDeviceToHost);

  hipFree(mat1d_GPU);
  hipFree(colsum_GPU);
}

void reduce_vector_gpu(double* vec, double* result, int n, struct Timer* timer) {
  // Compute execution GPU config
  dim3 dimBlock(1, 1);
  int blocks_in_grid = (int) ceil((double) n / BLOCK_SIZE);
  dim3 dimGrid(blocks_in_grid, 1);

  // Device: alloc
  double* vec_GPU;
  double* result_GPU;
  hipMalloc((void**) &vec_GPU, n*sizeof(double));
  hipMalloc((void**) &result_GPU, sizeof(double));

  // Host->Device copy
  hipMemcpy(vec_GPU, vec, n*sizeof(double), hipMemcpyHostToDevice);

  // Device: execution + timing
  start_timer(timer);
  reduce_vector_gpu_kernel<<<dimGrid, dimBlock>>>(vec_GPU, result_GPU, n);
  end_timer(timer);

  cuda_last_error_check("reduce_vector_gpu");

  // Device->Host copy
  hipMemcpy(vec, vec_GPU, n*sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(result, result_GPU, sizeof(double), hipMemcpyDeviceToHost);

  hipFree(result_GPU);
}


// Kernels

__global__ void add_rows_gpu_kernel(double* mat, double* out, int n, int m) {
       int x = blockIdx.x * BLOCK_SIZE + threadIdx.x;
       int y = threadIdx.y;
       if (x < n && y == 0) { // Only 0th thread in the y dimension is used
	 out[x] = 0;
	 for (int i = 0; i < m; i++) {
	   out[x] += mat[i+(x*m)];
	 }
       }
}

__global__ void add_cols_gpu_kernel(double* mat, double* out, int n, int m) {
       int x = threadIdx.x;
       int y = blockIdx.x * BLOCK_SIZE + threadIdx.y;
       if (y < m && x == 0) { // Only 0th thread in the x dimension is used
	 out[y] = 0;
	 for (int i = 0; i < n; i++) {
	   out[y] += mat[(i*m)+y];
	 }
       }
}

__global__ void reduce_vector_gpu_kernel(double* vec, double* result, int n) {
       int x = threadIdx.x;
       int y = threadIdx.y;
       if (x == 0 && y == 0) { // Only 1 thread used
	 *result = 0;
	 for (int i = 0; i < n; i++) {
	   *result += vec[i];
	 }
       }
}

// Cuda error check util
void cuda_last_error_check (const char *message) {
	hipError_t err = hipGetLastError();
	if(hipSuccess != err) {
		printf("[CUDA] [ERROR] %s: %s\n", message, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}
